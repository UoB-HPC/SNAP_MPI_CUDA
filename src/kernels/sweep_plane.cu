#include "hip/hip_runtime.h"

struct cell_id
{
    unsigned int i, j, k;
};


#define SOURCE_INDEX(m,g,i,j,k,cmom,ng,nx,ny) ((m)+((cmom)*(g))+((cmom)*(ng)*(i))+((cmom)*(ng)*(nx)*(j))+((cmom)*(ng)*(nx)*(ny)*(k)))
#define SCAT_COEFF_INDEX(a,l,o,nang,cmom) ((a)+((nang)*(l))+((nang)*(cmom)*o))
#define FLUX_I_INDEX(a,g,j,k,nang,ng,ny) ((a)+((nang)*(g))+((nang)*(ng)*(j))+((nang)*(ng)*(ny)*(k)))
#define FLUX_J_INDEX(a,g,i,k,nang,ng,nx) ((a)+((nang)*(g))+((nang)*(ng)*(i))+((nang)*(ng)*(nx)*(k)))
#define FLUX_K_INDEX(a,g,i,j,nang,ng,nx) ((a)+((nang)*(g))+((nang)*(ng)*(i))+((nang)*(ng)*(nx)*(j)))
#define ANGULAR_FLUX_INDEX(a,g,i,j,k,nang,ng,nx,ny) ((a)+((nang)*(g))+((nang)*(ng)*(i))+((nang)*(ng)*(nx)*(j))+((nang)*(ng)*(nx)*(ny)*(k)))
#define DENOMINATOR_INDEX(a,g,i,j,k,nang,ng,nx,ny) ((a)+((nang)*(g))+((nang)*(ng)*(i))+((nang)*(ng)*(nx)*(j))+((nang)*(ng)*(nx)*(ny)*(k)))

#define source(m,g,i,j,k) source[SOURCE_INDEX((m),(g),(i),(j),(k),cmom,ng,nx,ny)]
#define scat_coeff(a,l,o) scat_coeff[SCAT_COEFF_INDEX((a),(l),(o),nang,cmom)]
#define flux_i(a,g,j,k) flux_i[FLUX_I_INDEX((a),(g),(j),(k),nang,ng,ny)]
#define flux_j(a,g,i,k) flux_j[FLUX_J_INDEX((a),(g),(i),(k),nang,ng,nx)]
#define flux_k(a,g,i,j) flux_k[FLUX_K_INDEX((a),(g),(i),(j),nang,ng,nx)]
#define angular_flux_in(a,g,i,j,k) angular_flux_in[ANGULAR_FLUX_INDEX((a),(g),(i),(j),(k),nang,ng,nx,ny)]
#define angular_flux_out(a,g,i,j,k) angular_flux_out[ANGULAR_FLUX_INDEX((a),(g),(i),(j),(k),nang,ng,nx,ny)]
#define denominator(a,g,i,j,k) denominator[DENOMINATOR_INDEX((a),(g),(i),(j),(k),nang,ng,nx,ny)]


__global__ void sweep_plane(
    const unsigned int nx,
    const unsigned int ny,
    const unsigned int nz,
    const unsigned int nang,
    const unsigned int ng,
    const unsigned int cmom,
    const int istep,
    const int jstep,
    const int kstep,
    const unsigned int oct,
    const unsigned int z_pos,
    const struct cell_id * plane,
    const double * restrict source,
    const double * restrict scat_coeff,
    const double * restrict dd_i,
    const double * restrict dd_j,
    const double * restrict dd_k,
    const double * restrict mu,
    const double * restrict velocity_delta,
    const double * restrict mat_cross_section,
    const double * restrict denominator,
    const double * restrict angular_flux_in,
    double * restrict flux_i,
    double * restrict flux_j,
    double * restrict flux_k,
    double * restrict angular_flux_out
    )
{
    // Recover indexes for angle and group
    const size_t global_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t global_id_y = blockIdx.y * blockDim.y + threadIdx.y;

    const size_t a = global_id_x % nang;
    const size_t g = global_id_x / nang;

    // Read cell index from plane buffer
    const size_t i = (istep > 0) ? plane[global_id_y].i         : nx - plane[global_id_y].i         - 1;
    const size_t j = (jstep > 0) ? plane[global_id_y].j         : ny - plane[global_id_y].j         - 1;
    const size_t k = (kstep > 0) ? plane[global_id_y].k + z_pos : nz - plane[global_id_y].k - z_pos - 1;

    //
    // Compute the angular flux (psi)
    //

    // Begin with the first scattering moment
    double source_term = source(0,g,i,j,k);

    // Add in the anisotropic scattering source moments
    for (unsigned int l = 1; l < cmom; l++)
    {
        source_term += scat_coeff(a,l,oct) * source(l,g,i,j,k);
    }

    double psi =
        source_term
        + flux_i(a,g,j,k)*mu[a]*dd_i[0]
        + flux_j(a,g,i,k)*dd_j[a]
        + flux_k(a,g,i,j)*dd_k[a];

    // Add contribution from last timestep flux if time-dependant
    if (velocity_delta[g] != 0.0)
    {
        psi += velocity_delta[g] * angular_flux_in(a,g,i,j,k);
    }

    // "Divide" by denominator
    psi *= denominator(a,g,i,j,k);

    // Compute upwind fluxes
    double tmp_flux_i = 2.0 * psi - flux_i(a,g,j,k);
    double tmp_flux_j = 2.0 * psi - flux_j(a,g,i,k);
    double tmp_flux_k = 2.0 * psi - flux_k(a,g,i,j);

    // Time difference the final flux value
    if (velocity_delta[g] != 0.0)
    {
        psi = 2.0 * psi - angular_flux_in(a,g,i,j,k);
    }

    // Fixup
    double zeros[4];
    int num_ok = 4;
    for (int fix = 0; fix < 4; fix++)
    {
        zeros[0] = (tmp_flux_i < 0.0) ? 0.0 : 1.0;
        zeros[1] = (tmp_flux_j < 0.0) ? 0.0 : 1.0;
        zeros[2] = (tmp_flux_k < 0.0) ? 0.0 : 1.0;
        zeros[3] = (psi < 0.0)        ? 0.0 : 1.0;

        if (num_ok == zeros[0] + zeros[1] + zeros[2] + zeros[3])
            continue;

        num_ok = zeros[0] + zeros[1] + zeros[2] + zeros[3];

        // Recalculate psi
        psi =
            flux_i(a,g,j,k)*mu[a]*dd_i[0]*(1.0 + zeros[0]) +
            flux_j(a,g,i,k)*dd_j[a]*(1.0 + zeros[1]) +
            flux_k(a,g,i,j)*dd_k[a]*(1.0 + zeros[2]);

        if (velocity_delta[g] != 0.0)
        {
            psi += velocity_delta[g] * angular_flux_in(a,g,i,j,k) * (1.0 + zeros[3]);
        }

        psi = 0.5 * psi + source_term;

        double new_denominator =
            mat_cross_section[g] +
            mu[a] * dd_i[0] * zeros[0] +
            dd_j[a] * zeros[1] +
            dd_k[a] * zeros[2] +
            velocity_delta[g] * zeros[3];
        if (new_denominator > 1.0E-12)
        {
            psi /= new_denominator;
        }
        else
        {
            psi = 0.0;
        }

        tmp_flux_i = 2.0 * psi - flux_i(a,g,j,k);
        tmp_flux_j = 2.0 * psi - flux_j(a,g,i,k);
        tmp_flux_k = 2.0 * psi - flux_k(a,g,i,j);

        if (velocity_delta[g] != 0.0)
        {
            psi = 2.0 * psi - angular_flux_in(a,g,i,j,k);
        }

    }

    // Write values to global memory
    flux_i(a,g,j,k) = tmp_flux_i * zeros[0];
    flux_j(a,g,i,k) = tmp_flux_j * zeros[1];
    flux_k(a,g,i,j) = tmp_flux_k * zeros[2];
    angular_flux_out(a,g,i,j,k) = psi * zeros[3];
}
