#include "hip/hip_runtime.h"

#define SOURCE_INDEX(m,g,i,j,k,cmom,ng,nx,ny) ((m)+((cmom)*(g))+((cmom)*(ng)*(i))+((cmom)*(ng)*(nx)*(j))+((cmom)*(ng)*(nx)*(ny)*(k)))
#define FIXED_SOURCE_INDEX(g,i,j,k,ng,nx,ny) ((g)+((ng)*(i))+((ng)*(nx)*(j))+((ng)*(nx)*(ny)*(k)))
#define SCATTERING_MATRIX_INDEX(m,g1,g2,nmom,ng) ((m)+((nmom)*(g1))+((nmom)*(ng)*(g2)))
#define SCALAR_FLUX_INDEX(g,i,j,k,ng,nx,ny) ((g)+((ng)*(i))+((ng)*(nx)*(j))+((ng)*(nx)*(ny)*(k)))
#define SCALAR_FLUX_MOMENTS_INDEX(m,g,i,j,k,cmom,ng,nx,ny) ((m)+((cmom-1)*(g))+((cmom-1)*(ng)*(i))+((cmom-1)*(ng)*(nx)*(j))+((cmom-1)*(ng)*(nx)*(ny)*(k)))


#define outer_source(m,g,i,j,k) outer_source[SOURCE_INDEX((m),(g),(i),(j),(k),cmom,ng,nx,ny)]
#define fixed_source(g,i,j,k) fixed_source[FIXED_SOURCE_INDEX((g),(i),(j),(k),ng,nx,ny)]
#define scattering_matrix(m,g1,g2) scattering_matrix[SCATTERING_MATRIX_INDEX((m),(g1),(g2),nmom,ng)]
#define scalar_flux(g,i,j,k) scalar_flux[SCALAR_FLUX_INDEX((g),(i),(j),(k),ng,nx,ny)]
#define scalar_flux_moments(m,g,i,j,k) scalar_flux_moments[SCALAR_FLUX_MOMENTS_INDEX((m),(g),(i),(j),(k),cmom,ng,nx,ny)]

// 3D kernel, in local nx,ny,nz dimensions
// Probably not going to vectorise very well..
__global__ void calc_outer_source(
    const unsigned int nx,
    const unsigned int ny,
    const unsigned int nz,
    const unsigned int ng,
    const unsigned int cmom,
    const unsigned int nmom,
    const double * restrict fixed_source,
    const double * restrict scattering_matrix,
    const double * restrict scalar_flux,
    const double * restrict scalar_flux_moments,
    double * restrict outer_source
    )
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t k = blockIdx.z * blockDim.z + threadIdx.z;

    for (unsigned int g = 0; g < ng; g++)
    {
        // Set first moment to the fixed source
        outer_source(0,g,i,j,k) = fixed_source(g,i,j,k);

        // Loop over groups and moments to compute out-of-group scattering
        for (unsigned int g2 = 0; g2 < ng; g2++)
        {
            if (g == g2)
                continue;
            // Compute scattering source
            outer_source(0,g,i,j,k) += scattering_matrix(0,g2,g) * scalar_flux(g2,i,j,k);
            // Other moments
            unsigned int mom = 1;
            for (unsigned int l = 1; l < nmom; l++)
            {
                for (unsigned int m = 0; m < 2*l+1; m++)
                {
                    outer_source(mom,g,i,j,k) += scattering_matrix(l,g2,g) * scalar_flux_moments(mom-1,g2,i,j,k);
                    mom += 1;
                }
            }
        }
    }

}
