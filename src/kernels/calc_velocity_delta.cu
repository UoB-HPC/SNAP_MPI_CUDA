#include "hip/hip_runtime.h"

// Calculate the time absorbtion coefficient
__global__ void calc_velocity_delta(
    const double * restrict velocities,
    const double dt,
    double * restrict velocity_delta
    )
{
    size_t g = blockIdx.x * blockDim.x + threadIdx.x;
    velocity_delta[g] = 2.0 / (dt * velocities[g]);

}
