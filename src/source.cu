#include "hip/hip_runtime.h"

#include "source.h"


void compute_outer_source(
    const struct problem * problem,
    const struct rankinfo * rankinfo,
    struct buffers * buffers,
    struct events * events
    )
{
    dim3 grid(ceil(rankinfo->nx/(float)BLOCK_SIZE_2D), ceil(rankinfo->ny/(float)BLOCK_SIZE_2D), rankinfo->nz);
    dim3 threads(BLOCK_SIZE_2D, BLOCK_SIZE_2D, 1);

    hipEventRecord(events->outer_source_event_start);
    check_cuda("Recording outer source start event");

    calc_outer_source<<< grid, threads >>>(
        rankinfo->nx, rankinfo->ny, rankinfo->nz,
        problem->ng, problem->cmom, problem->nmom,
        buffers->fixed_source, buffers->scattering_matrix,
        buffers->scalar_flux, buffers->scalar_flux_moments,
        buffers->outer_source
    );
    check_cuda("Enqueue outer source kernel");

    hipEventRecord(events->outer_source_event_stop);
    check_cuda("Recording outer source stop event");
}


void compute_inner_source(
    const struct problem * problem,
    const struct rankinfo * rankinfo,
    struct buffers * buffers,
    struct events * events
    )
{
    dim3 grid(ceil(rankinfo->nx/(float)BLOCK_SIZE_2D), ceil(rankinfo->ny/(float)BLOCK_SIZE_2D), rankinfo->nz);
    dim3 threads(BLOCK_SIZE_2D, BLOCK_SIZE_2D, 1);

    hipEventRecord(events->inner_source_event_start);
    check_cuda("Recording inner source start event");

    calc_inner_source<<< grid, threads >>>(
        rankinfo->nx, rankinfo->ny, rankinfo->nz,
        problem->ng, problem->cmom, problem->nmom,
        buffers->outer_source, buffers->scattering_matrix,
        buffers->scalar_flux, buffers->scalar_flux_moments,
        buffers->inner_source
    );
    check_cuda("Enqueue inner source kernel");

    hipEventRecord(events->inner_source_event_stop);
    check_cuda("Recording inner source stop event");
}

