
#include "events.h"


void load_event(hipEvent_t e)
{
    hipEventRecord(e);
    check_cuda("Record event");
}

